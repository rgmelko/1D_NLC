#include "hip/hip_runtime.h"
#include "GPUmagnetization.h"

__host__ void GPUmagnetization( int HowMany, d_hamiltonian* & Ham, parameters* & data,  double** & Eigenvectors, double* & chi)
{
    int VecSize[ HowMany ];
    double** result = (double**)malloc( HowMany * sizeof(double*) ); 
    hipError_t status[ HowMany ];
    hipStream_t stream[ HowMany ];

    for( int i = 0; i < HowMany; i++)     
    {

        VecSize[i] = Ham[i].sectorDim;
        
        status[i] = hipStreamCreate( &stream[i] );
        if( status[i] != hipSuccess )
        {
            cout<<"Error creating "<<i<<"th stream in magnetization: "<<hipGetErrorString( status[i] )<<endl;
        }

        status[i] = hipMalloc( &result[i], VecSize[i]*sizeof(double));
        if( status[i] != hipSuccess )
        {
            cout<<"Error allocating "<<i<<"th result in magnetization: "<<hipGetErrorString( status[i] )<<endl;
        }

    }

    for( int i = 0; i < HowMany; i++)
    {
        MagnetSquared<<<VecSize[i]/512, 512, 0, stream[i]>>>(VecSize[i], data[i].nsite, Eigenvectors[i], result[i]);
    }


    for( int i = 0; i < HowMany; i++)
    {
        thrust::device_ptr<double> ReducePtr( result[i] );
        chi[ i ] = thrust::reduce(ReducePtr, ReducePtr + VecSize[i]);
    }
    
    for(int i = 0; i < HowMany; i++)
    {
        hipFree(result[i]);
        hipStreamDestroy(stream[i]);
    }
    free( VecSize );
    free( result );
}

__global__ void MagnetSquared( int VecSize, int SpinCount, double* Groundstate, double* MagSquared)
{
    int CurrentKet = threadIdx.x + (blockIdx.x * blockDim.x);
    int UpSpins = 0;

    if( CurrentKet < VecSize )
    {
        for( int CurrentSpin = 0; CurrentSpin < SpinCount; CurrentSpin++)
        {
            UpSpins += ( CurrentKet >> CurrentSpin) & 1;
        }
        MagSquared[ CurrentKet ] = ((2*UpSpins) - SpinCount ) * Groundstate[ CurrentKet ] * Groundstate[ CurrentKet ] * ((2*UpSpins) - SpinCount );
    }
}

