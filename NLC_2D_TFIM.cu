/*******************************************************************
Linked Cluster Expansion program for the 1D TFIM model
Roger Melko, Ann Kallin, Katie Hyatt June 2012
based on a Lanczos code from November 2007
********************************************************************/

using namespace std;

#include <fstream>
#include <limits.h>
#include <cstdio>
#include <time.h>
#include "CPU/Lanczos_07.h"
#include "CPU/GenHam.h"
#include "CPU/simparam.h"
#include "../CUDA/Lanczos/lanczos.h"
#include "graphs.h"

int main(int argc, char **argv) 
{

    double energy;

    PARAMS prm;  //Read parameters from param.dat  : see simparam.h
    double J;
    double h;

    vector< long double > eVec;

    J=prm.JJ_;
    h=prm.hh_;

    //ifstream fin("rectanglegraphs.dat");
    //bool TypeFlag = false;
    /*fin >> TypeFlag;
    fin.close();*/
    vector< graph > fileGraphs;
    vector< double > WeightHigh;

    ReadGraphsFromFile(fileGraphs, "rectanglegraphs.dat");

    //int HowMany = 30;

    ofstream fout("output_2D.dat");
    fout.precision(10);
    cout.precision(10);

    J=1;
    
    for(int hh=1; hh<10; hh++) 
    {
        h = hh;

        WeightHigh.push_back(-h); //Weight for site zero
        double RunningSumHigh = WeightHigh[0];

        d_hamiltonian* HamilLancz = (d_hamiltonian*)malloc(sizeof(d_hamiltonian));
        parameters* data = (parameters*)malloc(sizeof(parameters));
        double** groundstates = (double**)malloc(sizeof(double*));
        double** eigenvalues = (double**)malloc(sizeof(double*));
        eigenvalues[ 0 ] = (double*)malloc(sizeof(double));
        int* NumElem = (int*)malloc(sizeof(int));
        int** Bonds = (int**)malloc(sizeof(int*));
        
        unsigned int i = 1;

        while ( i<fileGraphs.size() )//&& fileGraphs.at(i).NumberSites < 14) //skip the zeroth graph
        {
            cout<<fileGraphs[i].NumberSites<<endl; 
            if ( (argv[0] == "--gpu" || argv[0] == "-g") && fileGraphs[i].NumberSites > 14)
            {

                Bonds[ 0 ] = (int*)malloc(sizeof(int)*3*fileGraphs[i].NumberSites);
                for(unsigned int k = 0; k < fileGraphs[i].NumberSites; k++)
                {
                    Bonds[ 0 ][ k ] = k;
                    Bonds[ 0 ][ k + fileGraphs[i].NumberSites ] = fileGraphs[i].AdjacencyList.at(2*k).second;
                    Bonds[ 0 ][ k + 2*fileGraphs[i].NumberSites ] = fileGraphs[i].AdjacencyList.at(2*k + 1).second;
                }
                    
                data[ 0 ].Sz = 0;
                data[ 0 ].dimension = 2;
                data[ 0 ].J1 = J;
                data[ 0 ].J2 = h;
                data[ 0 ].modelType = 2;
                ConstructSparseMatrix( 1, Bonds, HamilLancz, data, NumElem, 1);
                lanczos( 1, NumElem, HamilLancz, groundstates, eigenvalues, 200, 1, 1e-12);
                
                energy = eigenvalues[ 0 ][0];
            }
            
            else
            {
                GENHAM HV(fileGraphs[i].NumberSites, J, h, fileGraphs[i].AdjacencyList, fileGraphs[i].LowField);

                LANCZOS lancz(HV.Vdim);  //dimension of reduced Hilbert space (Sz sector)
                HV.SparseHamJQ();  //generates sparse matrix Hamiltonian for Lanczos
            
                energy = lancz.Diag(HV, 1, 1, eVec); // Hamiltonian, # of eigenvalues to converge, 1 for -values only, 2 for vals AND vectors
            }

            WeightHigh.push_back(energy);
            for (unsigned int j = 0; j < fileGraphs[i].SubgraphList.size(); j++)
                WeightHigh.back() -= fileGraphs[i].SubgraphList[j].second * WeightHigh[fileGraphs[i].SubgraphList[j].first];

            cout<<"h="<<h<<" J="<<J<<" graph #"<<i<<"  ";
            //        cout<<" energy "<<setprecision(12)<<energy<<endl;
            //        cout<<"WeightHigh["<<i<<"] = "<<WeightHigh.back()<<endl;
            RunningSumHigh += WeightHigh.back();
            cout<<"RunningSumHigh = "<<RunningSumHigh;
            cout<<endl;
            i++;
        } 
        /*
        if( argv[0] == "--gpu" || argv[0] == "-g" )
        {
            while ( i < fileGraphs.size() )
            {
                i += 30;
                if (fileGraphs.at(i).NumberSites == 18 )
                {
                    HowMany = 2;
                }
                for( int j = 0; j < HowMany; j++)
                {
                
                    Bonds[ j ] = (int*)malloc(sizeof(int)*3*fileGraphs.at(i - j).NumberSites);
                    for(unsigned int k = 0; k < fileGraphs.at(i - j).NumberSites; k++)
                    {
                        Bonds[ j ][ k ] = k;
                        Bonds[ j ][ k + fileGraphs.at(i - j).NumberSites ] = fileGraphs.at(i - j).AdjacencyList.at(2*k).second;
                        Bonds[ j ][ k + 2*fileGraphs.at(i - j).NumberSites ] = fileGraphs.at(i - j).AdjacencyList.at(2*k + 1).second;
                    }
                    
                    data[ j ].Sz = 0;
                    data[ j ].dimension = 2;
                    data[ j ].J1 = J;
                    data[ j ].J2 = h;
                    data[ j ].modelType = 2;
                    eigenvalues[ j ] = (double*)malloc(sizeof(double));
                }
                
                ConstructSparseMatrix(HowMany, Bonds, HamilLancz, data, NumElem, 1);
                lanczos(HowMany, NumElem, HamilLancz, groundstates, eigenvalues, 200, 1, 1e-12);
                
                for( int j = 0; j < HowMany; j++)
                {
                    energy = eigenvalues[ HowMany - 1 - j ][0];
                    WeightHigh.push_back(energy);
                    for( unsigned int k = 0; k < fileGraphs.at(i - j).SubgraphList.size(); k++)
                    {
                        WeightHigh.back() -= fileGraphs.at(i - j).SubgraphList[k].second * WeightHigh[fileGraphs.at(i - j).SubgraphList[k].first];

                        cout<<"h="<<h<<" J="<<J<<" graph #"<<i - j<<"  ";
                        //cout<<" energy "<<setprecision(12)<<energy<<endl;
                        //cout<<"WeightHigh["<<i<<"] = "<<WeightHigh.back()<<endl;
                        RunningSumHigh += WeightHigh.back();
                        cout <<"RunningSumHigh = "<< RunningSumHigh;
                        cout<<endl;
                    }
                    free(Bonds[j]);
                    hipFree(groundstates[j]);
                    hipFree(eigenvalues[j]);
                    hipFree(HamilLancz[j].rows);
                    hipFree(HamilLancz[j].cols);
                    hipFree(HamilLancz[j].vals);
                }
            }
        }    

        else 
        {
            while ( i < fileGraphs.size() )
            {
        //---High-Field---
                GENHAM HV(fileGraphs.at(i).NumberSites, J, h, fileGraphs.at(i).AdjacencyList, fileGraphs.at(i).LowField);

                LANCZOS lancz(HV.Vdim);  //dimension of reduced Hilbert space (Sz sector)
                HV.SparseHamJQ();  //generates sparse matrix Hamiltonian for Lanczos
                energy = lancz.Diag(HV, 1, prm.valvec_, eVec); // Hamiltonian, # of eigenvalues to converge, 1 for -values only, 2 for vals AND vectors
                WeightHigh.push_back(energy);
                for (unsigned int j = 0; j<fileGraphs.at(i).SubgraphList.size(); j++)
                    WeightHigh.back() -= fileGraphs.at(i).SubgraphList[j].second * WeightHigh[fileGraphs.at(i).SubgraphList[j].first];

                cout<<"h="<<h<<" J="<<J<<" graph #"<<i<<"  ";
                //cout<<" energy "<<setprecision(12)<<energy<<endl;
                //cout<<"WeightHigh["<<i<<"] = "<<WeightHigh.back()<<endl;
                RunningSumHigh += WeightHigh.back();
                cout <<"RunningSumHigh = "<< RunningSumHigh;
                cout<<endl;
                i++;
            }
        }*/

        fout<<"h= "<<h<<" J= "<<J;
        fout <<" Energy= "<< RunningSumHigh<< endl<<endl;

        WeightHigh.clear();
        RunningSumHigh=0;
    
    }
    
    fout.close();
    return 0;

}
