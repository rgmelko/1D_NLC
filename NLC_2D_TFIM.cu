/*******************************************************************
Linked Cluster Expansion program for the 1D TFIM model
Roger Melko, Ann Kallin, Katie Hyatt June 2012
based on a Lanczos code from November 2007
********************************************************************/

using namespace std;

#include <fstream>
#include <limits.h>
#include <cstdio>
#include <algorithm>
#include <time.h>
#include "CPU/Lanczos_07.h"
#include "CPU/GenHam.h"
#include "CPU/simparam.h"
#include "CPU/magnetization.h"
#include "GPU/GPUmagnetization.h"
#include "../Graphs/graphs.h"

#define HOW_MANY_16 30 
#define HOW_MANY_18 2 

bool order_16( Graph g ){ return g.Order == 16; };
bool order_18( Graph g ){ return g.Order == 18; };

int main(int argc, char **argv) 
{

    int CurrentArg = 1;
    bool gpuFlag = false;
    string InputFile;
    string OutputFile = "Output_2D.dat";
    while ( CurrentArg < argc )
    {
        if ( argv[ CurrentArg ] == string("-g") || argv[ CurrentArg ] == string("--gpu") )
        {
            gpuFlag = true;
        }
        if ( argv[ CurrentArg ] == string("-i") || argv[ CurrentArg ] == string("--input") )
        {
            InputFile = string(argv[ CurrentArg + 1 ]);
        }
        if ( argv[ CurrentArg ] == string("-o") || argv[ CurrentArg ] == string("--output") )
        {
            OutputFile = string(argv[ CurrentArg + 1 ]);
        }
        CurrentArg++;
    }

    double energy;
    double magnet;

    PARAMS prm;  //Read parameters from param.dat  : see simparam.h
    double J;
    double h;

    vector< long double > eVec;

    J=prm.JJ_;
    h=prm.hh_;

    //ifstream fin("rectanglegraphs.dat");
    //bool TypeFlag = false;
    /*fin >> TypeFlag;
    fin.close();*/
    vector< Graph > fileGraphs;
    vector< double > EnergyWeightHigh;
    vector< double > MagnetWeightHigh;

    ReadGraphsFromFile( fileGraphs, InputFile);

    //int HowMany = 30;

    ofstream fout( OutputFile.c_str() );
    fout.precision( 10 );
    cout.precision( 10 );

    int FirstCount; 
    FirstCount = (int) std::count_if(fileGraphs.begin(), fileGraphs.end(), order_16);
    int SecondCount; 
    SecondCount = (int) std::count_if(fileGraphs.begin(), fileGraphs.end(), order_18);

    J = 1;
    
    for( double hh = 4; hh <= 5; hh += 1 ) 
    {
        h = hh;

        EnergyWeightHigh.push_back( -h ); //Weight for site zero
        MagnetWeightHigh.push_back( 1. ); //Weight for site zero
        double EnergyRunningSumHigh = EnergyWeightHigh[ 0 ];
        double MagnetRunningSumHigh = MagnetWeightHigh[ 0 ];

        d_hamiltonian* HamilLancz = (d_hamiltonian*) malloc(HOW_MANY_16 * sizeof( d_hamiltonian ) );
        parameters* data          = (parameters*) malloc( HOW_MANY_16 * sizeof( parameters ) );
        double** groundstates     = (double**) malloc( HOW_MANY_16 * sizeof( double* ) );
        double** eigenvalues      = (double**) malloc( HOW_MANY_16 * sizeof( double* ) );
        double* magnetarray       = (double*) malloc( HOW_MANY_16 * sizeof(double* ) );
        int* NumElem = (int*) malloc( HOW_MANY_16 * sizeof( int ) );
        int** Bonds  = (int**) malloc( HOW_MANY_16 * sizeof( int* ) );
        unsigned int GPUqueue[HOW_MANY_16];

        int GPUprocessed = 0;
        int GPUmax = HOW_MANY_16;
        int remaining = FirstCount;

        unsigned int i = 1;
        while ( i < fileGraphs.size() )//&& fileGraphs.at(i).Order < 14) //skip the zeroth graph
        {
            if ( gpuFlag && 
                 GPUprocessed < GPUmax &&
                 remaining >= GPUmax &&
                 (fileGraphs[ i ].Order == 16 || fileGraphs[ i ].Order == 18) )
            {
                GPUprocessed++;
                remaining--;
                GPUqueue[GPUprocessed] = i; //store the locations of the graphs we're going to process in parallel

                //energy = 0;
                Bonds[ GPUprocessed ] = ( int* ) malloc( sizeof( int ) * 3 * fileGraphs[ i ].Order );
                for ( unsigned int k = 0; k < fileGraphs[ i ].Order; k++ )
                {
                    Bonds[ GPUprocessed ][ k ] = k;
                    Bonds[ GPUprocessed ][ k + fileGraphs[ i ].Order ] = fileGraphs[ i ].AdjacencyList[ k ].second;
                    Bonds[ GPUprocessed ][ k + 2 * fileGraphs[ i ].Order ] = fileGraphs[ i ].AdjacencyList[ 2 * k + 1 ].second;
                }
                    
                data[ GPUprocessed ].nsite = fileGraphs[ i ].Order;
                data[ GPUprocessed ].Sz = 0;
                data[ GPUprocessed ].dimension = (fileGraphs[i].AdjacencyList.size() <= fileGraphs[i].Order) ? 1 : 2;
                data[ GPUprocessed ].J1 = 4*J;
                data[ GPUprocessed ].J2 = h;
                data[ GPUprocessed ].modelType = 2;
                eigenvalues[ GPUprocessed ] = (double*) malloc( sizeof( double ) );
                i++;
                /*hipEvent_t start, stop;
                hipEventCreate(&start);
                hipEventCreate(&stop);
                float time;
                hipEventRecord(start, 0);
                */
            }
            
            if( GPUprocessed == GPUmax - 1 || remaining == 0)
            {
                ConstructSparseMatrix( GPUprocessed, Bonds, HamilLancz, data, NumElem, 0);
                lanczos( GPUprocessed, NumElem, HamilLancz, groundstates, eigenvalues, 200, 1, 1e-12);
                GPUmagnetization( GPUprocessed, HamilLancz, data, groundstates, magnetarray);
                /*hipEventRecord(stop, 0);
                hipEventSynchronize(stop);
                hipEventElapsedTime(&time, start, stop);
                cout<<"Time to do GPU work: "<<time<<endl;
                hipEventDestroy(start);
                hipEventDestroy(stop);
                */
                for( int j = 0; j < GPUprocessed; j++ )
                {
                    energy = eigenvalues[ j ][ 0 ];
                    magnet = magnetarray[ j ];
                    EnergyWeightHigh.push_back( energy );
                    MagnetWeightHigh.push_back( magnet );
                    
                    for ( unsigned int k = 0; k < fileGraphs[ GPUqueue[j] ].SubgraphList.size(); k++ )
                    {
                        EnergyWeightHigh.back() -= fileGraphs[ GPUqueue[j] ].SubgraphList[ k ].second * EnergyWeightHigh[ fileGraphs[ GPUqueue[i] ].SubgraphList[ k ].first ];
                        MagnetWeightHigh.back() -= fileGraphs[ GPUqueue[j] ].SubgraphList[ k ].second * MagnetWeightHigh[ fileGraphs[ GPUqueue[i] ].SubgraphList[ k ].first ];
                    }
                    EnergyRunningSumHigh += fileGraphs[ GPUqueue[j] ].LatticeConstant * EnergyWeightHigh.back();
                    MagnetRunningSumHigh += fileGraphs[ GPUqueue[j] ].LatticeConstant * MagnetWeightHigh.back();
                }
                GPUprocessed = 0;
                //free(Bonds[0]);
                //hipFree(HamilLancz[0].rows);
                //hipFree(HamilLancz[0].cols);
                //hipFree(HamilLancz[0].vals);
                //hipFree(eigenvalues[0]);
                //hipFree(groundstates[0]);
            }
            if ( remaining == 0 )
            {
                remaining = SecondCount;
                GPUmax = HOW_MANY_18;
            }
            
            else
            {
                GENHAM HV( fileGraphs[ i ].Order, J, h, fileGraphs[ i ].AdjacencyList, fileGraphs[ i ].LowField );

                LANCZOS lancz( HV.Vdim );  //dimension of reduced Hilbert space (Sz sector)
                HV.SparseHamJQ();  //generates sparse matrix Hamiltonian for Lanczos
            
                energy = lancz.Diag( HV, 1, 1, eVec ); // Hamiltonian, # of eigenvalues to converge, 1 for -values only, 2 for vals AND vectors
                magnet = Magnetization( eVec, fileGraphs[ i ].Order);
                EnergyWeightHigh.push_back( energy );
                MagnetWeightHigh.push_back( magnet );
                for ( unsigned int j = 0; j < fileGraphs[ i ].SubgraphList.size(); j++ )
                {
                    EnergyWeightHigh.back() -= fileGraphs[ i ].SubgraphList[ j ].second * EnergyWeightHigh[ fileGraphs[ i ].SubgraphList[ j ].first ];
                    MagnetWeightHigh.back() -= fileGraphs[ i ].SubgraphList[ j ].second * MagnetWeightHigh[ fileGraphs[ i ].SubgraphList[ j ].first ];
                }
                EnergyRunningSumHigh += fileGraphs[ i ].LatticeConstant * EnergyWeightHigh.back();
                MagnetRunningSumHigh += fileGraphs[ i ].LatticeConstant * MagnetWeightHigh.back();
                i++;
            }
        } 
        fout<<"h= "<<h<<" J= "<<J;
        fout <<" Energy= "<< EnergyRunningSumHigh<<" Magnetization= "<<MagnetRunningSumHigh<<endl;

        EnergyWeightHigh.clear();
        MagnetWeightHigh.clear();
        EnergyRunningSumHigh=0;
        MagnetRunningSumHigh=0;
    
    }
    
    fout.close();
    return 0;

}
