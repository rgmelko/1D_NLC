/*******************************************************************
Linked Cluster Expansion program for the 1D TFIM model
Roger Melko, Ann Kallin, Katie Hyatt June 2012
based on a Lanczos code from November 2007
********************************************************************/

using namespace std;

#include <fstream>
#include <limits.h>
#include <cstdio>
#include <time.h>
#include "CPU/Lanczos_07.h"
#include "CPU/GenHam.h"
#include "CPU/simparam.h"
#include "GPU/lanczos.h"
#include "graphs.h"

int main(int argc, char **argv) 
{

    int CurrentArg = 1;
    bool gpuFlag = false;
    string InputFile;
    string OutputFile = "Output_2D.dat";
    while ( CurrentArg < argc )
    {
        if ( argv[ CurrentArg ] == string("-g") || argv[ CurrentArg ] == string("--gpu") )
        {
            gpuFlag = true;
        }
        if ( argv[ CurrentArg ] == string("-i") || argv[ CurrentArg ] == string("--input") )
        {
            InputFile = string(argv[ CurrentArg + 1 ]);
        }
        if ( argv[ CurrentArg ] == string("-o") || argv[ CurrentArg ] == string("--output") )
        {
            OutputFile = string(argv[ CurrentArg + 1 ]);
        }
        CurrentArg++;
    }

    double energy;

    PARAMS prm;  //Read parameters from param.dat  : see simparam.h
    double J;
    double h;

    vector< long double > eVec;

    J=prm.JJ_;
    h=prm.hh_;

    //ifstream fin("rectanglegraphs.dat");
    //bool TypeFlag = false;
    /*fin >> TypeFlag;
    fin.close();*/
    vector< graph > fileGraphs;
    vector< double > WeightHigh;

    ReadGraphsFromFile( fileGraphs, InputFile);

    //int HowMany = 30;

    ofstream fout( OutputFile.c_str() );
    fout.precision( 10 );
    cout.precision( 10 );

    J = 1;
    
    for( int hh = 1; hh < 2; hh++ ) 
    {
        h = hh;

        WeightHigh.push_back( -h ); //Weight for site zero
        double RunningSumHigh = WeightHigh[ 0 ];

        d_hamiltonian* HamilLancz = (d_hamiltonian*) malloc( sizeof( d_hamiltonian ) );
        parameters* data = (parameters*) malloc( sizeof( parameters ) );
        double** groundstates = (double**) malloc( sizeof( double* ) );
        double** eigenvalues = (double**) malloc( sizeof( double* ) );
        eigenvalues[ 0 ] = (double*) malloc( sizeof( double ) );
        int* NumElem = (int*) malloc( sizeof( int ) );
        int** Bonds = (int**) malloc( sizeof( int* ) );
        
        unsigned int i = 1;
        while ( i < fileGraphs.size() )//&& fileGraphs.at(i).NumberSites < 14) //skip the zeroth graph
        {
            if ( gpuFlag && fileGraphs[ i ].NumberSites == 16 || fileGraphs[ i ].NumberSites == 18 )
            {
                Bonds[ 0 ] = ( int* ) malloc( sizeof( int ) * 3 * fileGraphs[ i ].NumberSites );
                for ( unsigned int k = 0; k < fileGraphs[ i ].NumberSites; k++ )
                {
                    Bonds[ 0 ][ k ] = k;
                    Bonds[ 0 ][ k + fileGraphs[ i ].NumberSites ] = fileGraphs[ i ].AdjacencyList[ k ].second;
                    Bonds[ 0 ][ k + 2 * fileGraphs[ i ].NumberSites ] = fileGraphs[ i ].AdjacencyList[ 2 * k + 1 ].second;
                }
                    
                data[ 0 ].nsite = fileGraphs[ i ].NumberSites;
                data[ 0 ].Sz = 0;
                data[ 0 ].dimension = (fileGraphs[i].AdjacencyList.size() <= fileGraphs[i].NumberSites) ? 1 : 2;
                data[ 0 ].J1 = 4*J;
                data[ 0 ].J2 = h;
                data[ 0 ].modelType = 2;
                ConstructSparseMatrix( 1, Bonds, HamilLancz, data, NumElem, 1);
                lanczos( 1, NumElem, HamilLancz, groundstates, eigenvalues, 200, 1, 1e-12);
                
                energy = eigenvalues[ 0 ][ 0 ];
                //free(Bonds[0]);
                //hipFree(HamilLancz[0].rows);
                //hipFree(HamilLancz[0].cols);
                //hipFree(HamilLancz[0].vals);
                //hipFree(eigenvalues[0]);
                //hipFree(groundstates[0]);
            }
            
            else
            {
                GENHAM HV( fileGraphs[ i ].NumberSites, J, h, fileGraphs[ i ].AdjacencyList, fileGraphs[ i ].LowField );

                LANCZOS lancz( HV.Vdim );  //dimension of reduced Hilbert space (Sz sector)
                HV.SparseHamJQ();  //generates sparse matrix Hamiltonian for Lanczos
            
                energy = lancz.Diag( HV, 1, 1, eVec ); // Hamiltonian, # of eigenvalues to converge, 1 for -values only, 2 for vals AND vectors
            }

            WeightHigh.push_back( energy );
            for ( unsigned int j = 0; j < fileGraphs[ i ].SubgraphList.size(); j++ )
                WeightHigh.back() -= fileGraphs[ i ].SubgraphList[ j ].second * WeightHigh[ fileGraphs[ i ].SubgraphList[ j ].first ];

            cout<<"h="<<h<<" J="<<J<<" graph #"<<i<<"  ";
            cout<<" energy "<<setprecision(12)<<energy<<endl;
            //        cout<<"WeightHigh["<<i<<"] = "<<WeightHigh.back()<<endl;
            RunningSumHigh += fileGraphs[ i ].LatticeConstant * WeightHigh.back();
            cout<<"RunningSumHigh = "<<RunningSumHigh;
            cout<<endl;
            i++;
        } 
        /*
        if( argv[0] == "--gpu" || argv[0] == "-g" )
        {
            while ( i < fileGraphs.size() )
            {
                i += 30;
                if (fileGraphs.at(i).NumberSites == 18 )
                {
                    HowMany = 2;
                }
                for( int j = 0; j < HowMany; j++)
                {
                
                    Bonds[ j ] = (int*)malloc(sizeof(int)*3*fileGraphs.at(i - j).NumberSites);
                    for(unsigned int k = 0; k < fileGraphs.at(i - j).NumberSites; k++)
                    {
                        Bonds[ j ][ k ] = k;
                        Bonds[ j ][ k + fileGraphs.at(i - j).NumberSites ] = fileGraphs.at(i - j).AdjacencyList.at(2*k).second;
                        Bonds[ j ][ k + 2*fileGraphs.at(i - j).NumberSites ] = fileGraphs.at(i - j).AdjacencyList.at(2*k + 1).second;
                    }
                    
                    data[ j ].Sz = 0;
                    data[ j ].dimension = 2;
                    data[ j ].J1 = J;
                    data[ j ].J2 = h;
                    data[ j ].modelType = 2;
                    eigenvalues[ j ] = (double*)malloc(sizeof(double));
                }
                
                ConstructSparseMatrix(HowMany, Bonds, HamilLancz, data, NumElem, 1);
                lanczos(HowMany, NumElem, HamilLancz, groundstates, eigenvalues, 200, 1, 1e-12);
                
                for( int j = 0; j < HowMany; j++)
                {
                    energy = eigenvalues[ HowMany - 1 - j ][0];
                    WeightHigh.push_back(energy);
                    for( unsigned int k = 0; k < fileGraphs.at(i - j).SubgraphList.size(); k++)
                    {
                        WeightHigh.back() -= fileGraphs.at(i - j).SubgraphList[k].second * WeightHigh[fileGraphs.at(i - j).SubgraphList[k].first];

                        cout<<"h="<<h<<" J="<<J<<" graph #"<<i - j<<"  ";
                        //cout<<" energy "<<setprecision(12)<<energy<<endl;
                        //cout<<"WeightHigh["<<i<<"] = "<<WeightHigh.back()<<endl;
                        RunningSumHigh += WeightHigh.back();
                        cout <<"RunningSumHigh = "<< RunningSumHigh;
                        cout<<endl;
                    }
                    free(Bonds[j]);
                    hipFree(groundstates[j]);
                    hipFree(eigenvalues[j]);
                    hipFree(HamilLancz[j].rows);
                    hipFree(HamilLancz[j].cols);
                    hipFree(HamilLancz[j].vals);
                }
            }
        }    

        else 
        {
            while ( i < fileGraphs.size() )
            {
        //---High-Field---
                GENHAM HV(fileGraphs.at(i).NumberSites, J, h, fileGraphs.at(i).AdjacencyList, fileGraphs.at(i).LowField);

                LANCZOS lancz(HV.Vdim);  //dimension of reduced Hilbert space (Sz sector)
                HV.SparseHamJQ();  //generates sparse matrix Hamiltonian for Lanczos
                energy = lancz.Diag(HV, 1, prm.valvec_, eVec); // Hamiltonian, # of eigenvalues to converge, 1 for -values only, 2 for vals AND vectors
                WeightHigh.push_back(energy);
                for (unsigned int j = 0; j<fileGraphs.at(i).SubgraphList.size(); j++)
                    WeightHigh.back() -= fileGraphs.at(i).SubgraphList[j].second * WeightHigh[fileGraphs.at(i).SubgraphList[j].first];

                cout<<"h="<<h<<" J="<<J<<" graph #"<<i<<"  ";
                //cout<<" energy "<<setprecision(12)<<energy<<endl;
                //cout<<"WeightHigh["<<i<<"] = "<<WeightHigh.back()<<endl;
                RunningSumHigh += WeightHigh.back();
                cout <<"RunningSumHigh = "<< RunningSumHigh;
                cout<<endl;
                i++;
            }
        }*/

        fout<<"h= "<<h<<" J= "<<J;
        fout <<" Energy= "<< RunningSumHigh<< endl<<endl;

        WeightHigh.clear();
        RunningSumHigh=0;
    
    }
    
    fout.close();
    return 0;

}
